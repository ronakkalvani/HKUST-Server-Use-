#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <vector>

#define BLOCK_THREADS 32
#define ITEMS_PER_THREAD 1

// Block-sorting CUDA kernel
__global__ void BlockSortKernel2(int *d_in, int *d_out, int *d_block_starts, int num_blocks, int num_elements)
{
    // Specialize BlockLoad, BlockStore, and BlockRadixSort collective types
    typedef hipcub::BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_VECTORIZE> BlockLoadT;
    typedef hipcub::BlockStore<int, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_STORE_VECTORIZE> BlockStoreT;
    typedef hipcub::BlockRadixSort<int, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;

    // Allocate type-safe, repurposable shared memory for collectives
    __shared__ union {
        typename BlockLoadT::TempStorage load;
        typename BlockStoreT::TempStorage store;
        typename BlockRadixSortT::TempStorage sort;
    } temp_storage;

    // Determine the range of elements this block should sort
    int block_start = d_block_starts[blockIdx.x];
    int block_end = (blockIdx.x == num_blocks - 1) ? num_elements : d_block_starts[blockIdx.x + 1];
    int num_items = block_end - block_start;

    // Ensure we do not access out of bounds memory
    if (block_start >= num_elements) return;

    // Load data
    int thread_keys[ITEMS_PER_THREAD];
    int items_to_load = min(num_items - threadIdx.x * ITEMS_PER_THREAD, ITEMS_PER_THREAD);
    BlockLoadT(temp_storage.load).Load(d_in + block_start + threadIdx.x * ITEMS_PER_THREAD, thread_keys, items_to_load);

    __syncthreads(); // Barrier for smem reuse

    // Collectively sort the keys
    BlockRadixSortT(temp_storage.sort).Sort(thread_keys);

    __syncthreads(); // Barrier for smem reuse

    // Store the sorted segment
    BlockStoreT(temp_storage.store).Store(d_out + block_start + threadIdx.x * ITEMS_PER_THREAD, thread_keys, items_to_load);
}

int main() {
    // Initialize host data
    std::vector<int> h_data(1024);
    for (int i = 0; i < h_data.size(); i++) {
        h_data[i] = rand() % 127;
        std::cout << h_data[i] << " ";
    }
    std::cout << "\n";
    int n = h_data.size();

    // Define block starting indices
    std::vector<int> h_block_starts(n / BLOCK_THREADS);
    for (int i = 0; i < n / BLOCK_THREADS; i++) {
        if (i % 2) h_block_starts[i] = (i) * (BLOCK_THREADS) + 7;
        else h_block_starts[i] = (i) * (BLOCK_THREADS);
        std::cout << h_block_starts[i] << " ";
    }
    std::cout << "\n";
    int num_blocks = h_block_starts.size();

    // Allocate device memory
    int *d_data, *d_sorted_data, *d_block_starts;
    hipMalloc(&d_data, n * sizeof(int));
    hipMalloc(&d_sorted_data, n * sizeof(int));
    hipMalloc(&d_block_starts, num_blocks * sizeof(int));

    // Copy data to device
    hipMemcpy(d_data, h_data.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_block_starts, h_block_starts.data(), num_blocks * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel to sort blocks
    BlockSortKernel2<<<num_blocks, BLOCK_THREADS>>>(d_data, d_sorted_data, d_block_starts, num_blocks, n);

    // Copy sorted data back to host
    hipMemcpy(h_data.data(), d_sorted_data, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print sorted blocks
    for (int i = 0; i < h_data.size(); i++) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_data);
    hipFree(d_sorted_data);
    hipFree(d_block_starts);

    return 0;
}

// #include <hip/hip_runtime.h>
// #include <hipcub/hipcub.hpp>
// #include <iostream>
// #include <vector>

// #define BLOCK_THREADS 32
// #define ITEMS_PER_THREAD 1

// // Block-sorting CUDA kernel
// __global__ void BlockSortKernel2(int *d_in, int *d_out, int *d_block_starts, int num_blocks, int num_elements)
// {
//     // Specialize BlockLoad, BlockStore, and BlockRadixSort collective types
//     typedef hipcub::BlockLoad<int, 2*BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_VECTORIZE> BlockLoadT;
//     typedef hipcub::BlockStore<int, 2*BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_STORE_VECTORIZE> BlockStoreT;
//     typedef hipcub::BlockRadixSort<int, 2*BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;

//     // Allocate type-safe, repurposable shared memory for collectives
//     __shared__ union {
//         typename BlockLoadT::TempStorage load;
//         typename BlockStoreT::TempStorage store;
//         typename BlockRadixSortT::TempStorage sort;
//     } temp_storage;

//     // Determine the range of elements this block should sort
//     int block_start = d_block_starts[blockIdx.x];
//     int block_end = (blockIdx.x == num_blocks - 1) ? num_elements : d_block_starts[blockIdx.x + 1];
//     int num_items = block_end - block_start;

//     // Ensure we do not access out of bounds memory
//     if (block_start >= num_elements) return;

//     // Load data
//     int thread_keys[ITEMS_PER_THREAD];
//     int valid_items = min(num_items - threadIdx.x * ITEMS_PER_THREAD, ITEMS_PER_THREAD);
//     BlockLoadT(temp_storage.load).Load(d_in + block_start, thread_keys, num_items);

//     __syncthreads(); // Barrier for smem reuse

//     // Collectively sort the keys
//     BlockRadixSortT(temp_storage.sort).Sort(thread_keys);

//     __syncthreads(); // Barrier for smem reuse

//     // Store the sorted segment
//     BlockStoreT(temp_storage.store).Store(d_out + block_start, thread_keys, num_items);
    
// }


// int main() {
//     // Initialize host data
//     std::vector<int> h_data(1024);
//     for (int i = 0; i < h_data.size(); i++) {
//         h_data[i] = rand() % 127;
//         std::cout<<h_data[i]<<" ";
//     }
//     std::cout<<"\n";
//     int n = h_data.size();

//     // Define block starting indices
//     // std::vector<int> h_block_starts = {0, 1000, 2000, 3000, 4000, 4500}; // Example block starts
//     std::vector<int> h_block_starts(n/BLOCK_THREADS);
//     for(int i=0;i<n/BLOCK_THREADS;i++) {
//         if (i%2) h_block_starts[i] = (i)*(BLOCK_THREADS)+7;
//         else h_block_starts[i] = (i)*(BLOCK_THREADS);
//         std::cout<<h_block_starts[i]<<" ";
//     }
//     std::cout<<"\n";
//     int num_blocks = h_block_starts.size();

//     // Allocate device memory
//     int *d_data, *d_sorted_data, *d_block_starts;
//     hipMalloc(&d_data, n * sizeof(int));
//     hipMalloc(&d_sorted_data, n * sizeof(int));
//     hipMalloc(&d_block_starts, num_blocks * sizeof(int));

//     // Copy data to device
//     hipMemcpy(d_data, h_data.data(), n * sizeof(int), hipMemcpyHostToDevice);
//     hipMemcpy(d_block_starts, h_block_starts.data(), num_blocks * sizeof(int), hipMemcpyHostToDevice);

//     // Launch kernel to sort blocks
//     BlockSortKernel2<<<num_blocks, BLOCK_THREADS*2>>>(d_data, d_sorted_data, d_block_starts, num_blocks, n);

//     // Copy sorted data back to host
//     hipMemcpy(h_data.data(), d_sorted_data, n * sizeof(int), hipMemcpyDeviceToHost);

//     // Print sorted blocks
//     for (int i = 0; i < h_data.size(); i++) {
//         std::cout << h_data[i] << " ";
//     }
//     std::cout << std::endl;

//     // Free device memory
//     hipFree(d_data);
//     hipFree(d_sorted_data);
//     hipFree(d_block_starts);

//     return 0;
// }
