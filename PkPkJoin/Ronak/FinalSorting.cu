#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <vector>

// Define the number of threads per block and items per thread
#define BLOCK_THREADS 128
#define ITEMS_PER_THREAD 4

// Block-sorting CUDA kernel
__global__ void BlockSortKernel(int *d_in, int *d_out, int num_blocks, int *block_indices)
{
    // Specialize BlockLoad, BlockStore, and BlockRadixSort collective types
    typedef hipcub::BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_TRANSPOSE> BlockLoadT;
    typedef hipcub::BlockStore<int, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_STORE_TRANSPOSE> BlockStoreT;
    typedef hipcub::BlockRadixSort<int, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;

    // Allocate type-safe, repurposable shared memory for collectives
    __shared__ union {
        typename BlockLoadT::TempStorage load;
        typename BlockStoreT::TempStorage store;
        typename BlockRadixSortT::TempStorage sort;
    } temp_storage;

    // Obtain this block's segment of consecutive keys (blocked across threads)
    int thread_keys[ITEMS_PER_THREAD];
    int block_idx = blockIdx.x;
    int block_start = block_indices[block_idx];
    int block_end = (block_idx + 1 < num_blocks) ? block_indices[block_idx + 1] : num_elements;
    int block_size = block_end - block_start;
    int valid_items = min(block_size, BLOCK_THREADS * ITEMS_PER_THREAD);

    // Initialize thread_keys with a known value for safer debugging
    for (int i = 0; i < ITEMS_PER_THREAD; i++) {
        thread_keys[i] = (block_start + threadIdx.x * ITEMS_PER_THREAD + i) < block_end ? d_in[block_start + threadIdx.x * ITEMS_PER_THREAD + i] : INT_MAX;
    }

    // Load data
    BlockLoadT(temp_storage.load).Load(d_in + block_start, thread_keys, valid_items);

    __syncthreads(); // Barrier for smem reuse

    // Collectively sort the keys
    BlockRadixSortT(temp_storage.sort).Sort(thread_keys);

    __syncthreads(); // Barrier for smem reuse

    // Store the sorted segment
    BlockStoreT(temp_storage.store).Store(d_out + block_start, thread_keys, valid_items);
}

int main() {
    // Initialize host data
    std::vector<int> h_data = {34, 78, 12, 56, 89, 21, 90, 34, 23, 45, 67, 11, 23, 56, 78, 99, 123, 45, 67, 89, 23, 45, 67, 34, 78};
    int n = h_data.size();

    // Define block start indices
    std::vector<int> h_block_indices = {0, 5, 10, 15, 20};

    // Allocate device memory
    int* d_data;
    hipMalloc(&d_data, n * sizeof(int));
    int* d_sorted_data;
    hipMalloc(&d_sorted_data, n * sizeof(int));
    int* d_block_indices;
    hipMalloc(&d_block_indices, h_block_indices.size() * sizeof(int));

    // Copy data to device
    hipMemcpy(d_data, h_data.data(), n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_block_indices, h_block_indices.data(), h_block_indices.size() * sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = h_block_indices.size();

    // Launch kernel to sort blocks
    BlockSortKernel<<<numBlocks, BLOCK_THREADS>>>(d_data, d_sorted_data, numBlocks, d_block_indices);

    // Copy sorted data back to host
    hipMemcpy(h_data.data(), d_sorted_data, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print sorted blocks
    for (int i = 0; i < h_data.size(); i++) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_data);
    hipFree(d_sorted_data);
    hipFree(d_block_indices);

    return 0;
}

// #include <hip/hip_runtime.h>
// #include <hipcub/hipcub.hpp>
// #include <iostream>
// #include <vector>

// #define BLOCK_THREADS 32
// #define ITEMS_PER_THREAD 1

// // Block-sorting CUDA kernel
// __global__ void BlockSortKernel2(int *d_in, int *d_out, int *d_block_starts, int num_blocks, int num_elements)
// {
//     // Specialize BlockLoad, BlockStore, and BlockRadixSort collective types
//     typedef hipcub::BlockLoad<int, 2*BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_VECTORIZE> BlockLoadT;
//     typedef hipcub::BlockStore<int, 2*BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_STORE_VECTORIZE> BlockStoreT;
//     typedef hipcub::BlockRadixSort<int, 2*BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;

//     // Allocate type-safe, repurposable shared memory for collectives
//     __shared__ union {
//         typename BlockLoadT::TempStorage load;
//         typename BlockStoreT::TempStorage store;
//         typename BlockRadixSortT::TempStorage sort;
//     } temp_storage;

//     // Determine the range of elements this block should sort
//     int block_start = d_block_starts[blockIdx.x];
//     int block_end = (blockIdx.x == num_blocks - 1) ? num_elements : d_block_starts[blockIdx.x + 1];
//     int num_items = block_end - block_start;

//     // Ensure we do not access out of bounds memory
//     if (block_start >= num_elements) return;

//     // Load data
//     int thread_keys[ITEMS_PER_THREAD];
//     int valid_items = min(num_items - threadIdx.x * ITEMS_PER_THREAD, ITEMS_PER_THREAD);
//     BlockLoadT(temp_storage.load).Load(d_in + block_start, thread_keys, num_items);

//     __syncthreads(); // Barrier for smem reuse

//     // Collectively sort the keys
//     BlockRadixSortT(temp_storage.sort).Sort(thread_keys);

//     __syncthreads(); // Barrier for smem reuse

//     // Store the sorted segment
//     BlockStoreT(temp_storage.store).Store(d_out + block_start, thread_keys, num_items);
    
// }


// int main() {
//     // Initialize host data
//     std::vector<int> h_data(1024);
//     for (int i = 0; i < h_data.size(); i++) {
//         h_data[i] = rand() % 127;
//         std::cout<<h_data[i]<<" ";
//     }
//     std::cout<<"\n";
//     int n = h_data.size();

//     // Define block starting indices
//     // std::vector<int> h_block_starts = {0, 1000, 2000, 3000, 4000, 4500}; // Example block starts
//     std::vector<int> h_block_starts(n/BLOCK_THREADS);
//     for(int i=0;i<n/BLOCK_THREADS;i++) {
//         if (i%2) h_block_starts[i] = (i)*(BLOCK_THREADS)+7;
//         else h_block_starts[i] = (i)*(BLOCK_THREADS);
//         std::cout<<h_block_starts[i]<<" ";
//     }
//     std::cout<<"\n";
//     int num_blocks = h_block_starts.size();

//     // Allocate device memory
//     int *d_data, *d_sorted_data, *d_block_starts;
//     hipMalloc(&d_data, n * sizeof(int));
//     hipMalloc(&d_sorted_data, n * sizeof(int));
//     hipMalloc(&d_block_starts, num_blocks * sizeof(int));

//     // Copy data to device
//     hipMemcpy(d_data, h_data.data(), n * sizeof(int), hipMemcpyHostToDevice);
//     hipMemcpy(d_block_starts, h_block_starts.data(), num_blocks * sizeof(int), hipMemcpyHostToDevice);

//     // Launch kernel to sort blocks
//     BlockSortKernel2<<<num_blocks, BLOCK_THREADS*2>>>(d_data, d_sorted_data, d_block_starts, num_blocks, n);

//     // Copy sorted data back to host
//     hipMemcpy(h_data.data(), d_sorted_data, n * sizeof(int), hipMemcpyDeviceToHost);

//     // Print sorted blocks
//     for (int i = 0; i < h_data.size(); i++) {
//         std::cout << h_data[i] << " ";
//     }
//     std::cout << std::endl;

//     // Free device memory
//     hipFree(d_data);
//     hipFree(d_sorted_data);
//     hipFree(d_block_starts);

//     return 0;
// }
