#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

// Error checking macro
#define CUDA_CHECK(call)                                                   \
    do {                                                                   \
        hipError_t error = call;                                          \
        if (error != hipSuccess) {                                        \
            std::cerr << "CUDA Error: " << hipGetErrorString(error) <<    \
            " at " << __FILE__ << ":" << __LINE__ << std::endl;            \
            exit(1);                                                       \
        }                                                                  \
    } while (0)

// Kernel to print array
__global__ void printArray(int* arr, int size) {
    for (int i = 0; i < size; ++i) {
        printf("%d ", arr[i]);
    }
    printf("\n");
}

__global__ void mergePartitions(
    int* d_subarrays, int* d_partitions, int* d_output, int* d_pivots, 
    int* d_partition_counts, int n, int p) 
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Step 1: Determine the partition for each element
    if (tid < n) {
        int partition = 0;
        while (partition < p - 1 && d_subarrays[tid] > d_pivots[partition]) {
            partition++;
        }

        printf("Element: %d, Partition: %d\n",tid,partition);

        // Step 2: Count the number of elements in each partition
        atomicAdd(&d_partition_counts[partition], 1);
    }

    // Synchronize threads to ensure all counts are computed
    __syncthreads();

    // Step 3: Compute the starting index for each partition
    if (tid == 0) {
        for (int i = 0; i < p; ++i) {
            printf("%d ", d_partition_counts[i]);
        }
        printf("\n");
        int sum = 0;
        for (int i = 0; i < p; ++i) {
            int temp = d_partition_counts[i];
            d_partition_counts[i] = sum;
            sum += temp;
        }
        for (int i = 0; i < p; ++i) {
            printf("%d ", d_partition_counts[i]);
        }
        printf("\n");
    }

    // Synchronize threads to ensure starting indices are computed
    __syncthreads();

    // Step 4: Distribute elements to the output array
    if (tid < n) {
        int partition = 0;
        while (partition < p - 1 && d_subarrays[tid] > d_pivots[partition]) {
            partition++;
        }
        int pos = atomicAdd(&d_partition_counts[partition], 1);
        d_output[pos] = d_subarrays[tid];
    }
}

int main() {
    // Example data
    // int h_subarrays[] = {1, 3, 5, 7, 2, 4, 6, 10, 8, 9, 11, 12};
    // int h_pivots[] = {4,8};

    const int n = 512;
    int p = 8;
    int h_subarrays[n];
    int h_pivots[p-1];
    for (int i=0;i<n;i++) {
        h_subarrays[i] = i % n;
    }
    for (int i=0;i<p-1;i++) {
        h_pivots[i] = (i+1)*(n/p);
    }

    // Device pointers
    int *d_subarrays, *d_output, *d_pivots, *d_partition_counts;

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_subarrays, n * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_output, n * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_pivots, (p - 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_partition_counts, p * sizeof(int)));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_subarrays, h_subarrays, n * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_pivots, h_pivots, (p - 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_partition_counts, 0, p * sizeof(int)));
    hipMemset(d_partition_counts, 0, p * sizeof(int));

    // Kernel launch parameters
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Launch kernel to merge partitions
    mergePartitions<<<numBlocks, blockSize>>>(d_subarrays, d_partition_counts, d_output, d_pivots, d_partition_counts, n, p);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    printArray<<<1,1>>>(d_partition_counts,p);

    // Copy result back to host
    int* h_output = new int[n];
    CUDA_CHECK(hipMemcpy(h_output, d_output, n * sizeof(int), hipMemcpyDeviceToHost));

    // Print result
    for (int i = 0; i < n; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    CUDA_CHECK(hipFree(d_subarrays));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_pivots));
    CUDA_CHECK(hipFree(d_partition_counts));

    delete[] h_output;

    return 0;
}
