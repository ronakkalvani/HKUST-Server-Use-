#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

// CUDA kernel to merge sorted blocks into a single sorted array
__global__ void mergeSortedBlocks(int* sorted_data, int* block_offsets, int* global_splitters, int num_blocks) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int block_start = block_offsets[blockIdx.x];
    int block_end = (blockIdx.x < num_blocks - 1) ? block_offsets[blockIdx.x + 1] : gridDim.x * blockDim.x;

    // Each thread merges its assigned block into the global sorted array
    for (int i = block_start + tid; i < block_end; i += blockDim.x) {
        // Perform binary search to find the correct position in global sorted array
        int value = sorted_data[i];
        int low = 0, high = num_blocks;
        while (low < high) {
            int mid = low + (high - low) / 2;
            if (value < global_splitters[mid]) {
                high = mid;
            } else {
                low = mid + 1;
            }
        }
        
        int insert_position = low;  // Position in the global sorted array
        // Perform atomic insertion (using CUDA atomicCAS for simplicity)
        int old = atomicCAS(&sorted_data[insert_position], sorted_data[insert_position], value);
        while (old != sorted_data[insert_position]) {
            old = atomicCAS(&sorted_data[insert_position], sorted_data[insert_position], value);
        }
    }
}

int main() {
    const int num_blocks = 4;  // Number of sorted blocks
    const int block_size = 256;  // Size of each sorted block
    const int total_size = num_blocks * block_size;

    // Initialize example sorted data
    int sorted_data[total_size];
    for (int i = 0; i < total_size; ++i) {
        sorted_data[i] = i;
    }

    // Example block offsets (starting index of each block in sorted_data)
    int block_offsets[num_blocks];
    for (int i = 0; i < num_blocks; ++i) {
        block_offsets[i] = i * block_size;
    }

    // Example global splitters (p-1 values that partition the data)
    int global_splitters[num_blocks - 1];
    for (int i = 0; i < num_blocks - 1; ++i) {
        global_splitters[i] = (i + 1) * (block_size - 1);  // Just an example, should be adjusted based on your actual data
    }

    // Print initial sorted data (example)
    std::cout << "Initial Sorted Data:" << std::endl;
    for (int i = 0; i < total_size; ++i) {
        std::cout << sorted_data[i] << " ";
    }
    std::cout << std::endl;

    // Initialize CUDA variables
    int* d_sorted_data;
    int* d_block_offsets;
    int* d_global_splitters;

    // Allocate memory on device
    hipMalloc((void**)&d_sorted_data, total_size * sizeof(int));
    hipMalloc((void**)&d_block_offsets, num_blocks * sizeof(int));
    hipMalloc((void**)&d_global_splitters, (num_blocks - 1) * sizeof(int));

    // Copy data to device memory
    hipMemcpy(d_sorted_data, sorted_data, total_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_block_offsets, block_offsets, num_blocks * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_global_splitters, global_splitters, (num_blocks - 1) * sizeof(int), hipMemcpyHostToDevice);

    // Launch CUDA kernel to merge sorted blocks
    int threads_per_block = 256;
    int blocks_per_grid = num_blocks;
    mergeSortedBlocks<<<blocks_per_grid, threads_per_block>>>(d_sorted_data, d_block_offsets, d_global_splitters, num_blocks);

    // Copy sorted data back to host
    hipMemcpy(sorted_data, d_sorted_data, total_size * sizeof(int), hipMemcpyDeviceToHost);

    // Print sorted data (after merging)
    std::cout << "Sorted Data:" << std::endl;
    for (int i = 0; i < total_size; ++i) {
        std::cout << sorted_data[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_sorted_data);
    hipFree(d_block_offsets);
    hipFree(d_global_splitters);

    return 0;
}
