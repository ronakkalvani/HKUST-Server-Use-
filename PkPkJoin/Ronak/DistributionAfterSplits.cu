#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

// Error checking macro
#define CUDA_CHECK(call)                                                   \
    do {                                                                   \
        hipError_t error = call;                                          \
        if (error != hipSuccess) {                                        \
            std::cerr << "CUDA Error: " << hipGetErrorString(error) <<    \
            " at " << __FILE__ << ":" << __LINE__ << std::endl;            \
            exit(1);                                                       \
        }                                                                  \
    } while (0)

// Kernel to print array
__global__ void printArray(int* arr, int size) {
    for (int i = 0; i < size; ++i) {
        printf("%d ", arr[i]);
    }
    printf("\n");
}

__global__ void countElements(
    int* d_subarrays, int* d_pivots, int* d_partition_counts, int n, int p) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        int partition = 0;
        while (partition < p - 1 && d_subarrays[tid] > d_pivots[partition]) {
            partition++;
        }
        atomicAdd(&d_partition_counts[partition], 1);
    }
}

__global__ void computeStarts(int* d_partition_counts, int* d_partition_starts, int p) {
    int tid = threadIdx.x;

    if (tid == 0) {
        int sum = 0;
        for (int i = 0; i < p; ++i) {
            d_partition_starts[i] = sum;
            sum += d_partition_counts[i];
        }
    }
}

__global__ void distributeElements(
    int* d_subarrays, int* d_output, int* d_pivots, 
    int* d_partition_starts, int* d_partition_offsets, int n, int p) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        int partition = 0;
        while (partition < p - 1 && d_subarrays[tid] > d_pivots[partition]) {
            partition++;
        }
        int pos = atomicAdd(&d_partition_offsets[partition], 1);
        d_output[d_partition_starts[partition] + pos] = d_subarrays[tid];
    }
}

int main() {
    const int n = 1e6;
    int p = n/(1024);
    int h_subarrays[n];
    int h_pivots[p-1];
    for (int i = 0; i < n; i++) {
        h_subarrays[i] = rand() % 12715;
    }
    for (int i = 0; i < p-1; i++) {
        h_pivots[i] = (i + 1) * (12715 / p);
        std::cout<<h_pivots[i]<<" ";
    }
    std::cout<<"\n";

    // Device pointers
    int *d_subarrays, *d_output, *d_pivots, *d_partition_counts, *d_partition_starts, *d_partition_offsets;

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_subarrays, n * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_output, n * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_pivots, (p - 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_partition_counts, p * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_partition_starts, p * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_partition_offsets, p * sizeof(int)));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_subarrays, h_subarrays, n * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_pivots, h_pivots, (p - 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemset(d_partition_counts, 0, p * sizeof(int)));
    CUDA_CHECK(hipMemset(d_partition_starts, 0, p * sizeof(int)));
    CUDA_CHECK(hipMemset(d_partition_offsets, 0, p * sizeof(int)));

    // Kernel launch parameters
    int blockSize = n/p;
    int numBlocks = (n + blockSize - 1) / blockSize;

    // Launch kernels in sequence to ensure synchronization
    countElements<<<numBlocks, blockSize>>>(d_subarrays, d_pivots, d_partition_counts, n, p);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    computeStarts<<<1, 1>>>(d_partition_counts, d_partition_starts, p);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    distributeElements<<<numBlocks, blockSize>>>(d_subarrays, d_output, d_pivots, d_partition_starts, d_partition_offsets, n, p);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    int* h_output = new int[n];
    CUDA_CHECK(hipMemcpy(h_output, d_output, n * sizeof(int), hipMemcpyDeviceToHost));

    // Print result
    for (int i = 0; i < n; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    CUDA_CHECK(hipFree(d_subarrays));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_pivots));
    CUDA_CHECK(hipFree(d_partition_counts));
    CUDA_CHECK(hipFree(d_partition_starts));
    CUDA_CHECK(hipFree(d_partition_offsets));

    delete[] h_output;

    return 0;
}


