#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
// #include <hip/hip_runtime.h>
// #include <hipcub/hipcub.hpp>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void findSplitsKernel(const int *data, int *output, const int *splitters, int numData, int numSplitters) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < numData) {
        // Perform binary search to find the appropriate partition
        int item = data[tid];
        int left = 0;
        int right = numSplitters - 1;
        while (left < right) {
            int mid = (left + right) / 2;
            if (item >= splitters[mid]) {
                left = mid;
            } else {
                right = mid - 1;
            }
        }
        output[tid] = left;  // 'left' is the partition index
    }
}
int main() {
    const int numData = 1e6;
    const int numSplitters = 1e3;

    // Example data and splitters
    int h_data[numData];
    int h_splitters[numSplitters];
    for(int i =0;i<numData;i++)
    {
        h_data[i] = rand()%1232443;
    }
    for(int i =0;i<numSplitters;i++)
    {
        h_data[i] = 1000*(i+1);
    }
    int h_output[numData];

    // Allocate device memory
    int *d_data, *d_splitters, *d_output;
    hipMalloc(&d_data, numData * sizeof(int));
    hipMalloc(&d_splitters, numSplitters * sizeof(int));
    hipMalloc(&d_output, numData * sizeof(int));

    // Copy data to device
    hipMemcpy(d_data, h_data, numData * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_splitters, h_splitters, numSplitters * sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numData + threadsPerBlock - 1) / threadsPerBlock;
    findSplitsKernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_output, d_splitters, numData, numSplitters);

    // Copy result back to host
    hipMemcpy(h_output, d_output, numData * sizeof(int), hipMemcpyDeviceToHost);

    // Display the results
    for (int i = 0; i < numData; ++i) {
        std::cout << "Data: " << h_data[i] << " -> Partition: " << h_output[i] << std::endl;
    }

    // Free device memory
    hipFree(d_data);
    hipFree(d_splitters);
    hipFree(d_output);

    return 0;
}

// Error checking macro
#define CUDA_CHECK(call)                                                   \
    do {                                                                   \
        hipError_t error = call;                                          \
        if (error != hipSuccess) {                                        \
            std::cerr << "CUDA Error: " << hipGetErrorString(error) <<    \
            " at " << __FILE__ << ":" << __LINE__ << std::endl;            \
            exit(1);                                                       \
        }                                                                  \
    } while (0)

// Kernel to print array
__global__ void printArray(int* arr, int size) {
    for (int i = 0; i < size; ++i) {
        printf("%d ", arr[i]);
    }
    printf("\n");
}

__global__ void countElements(
    int* d_subarrays, int* d_pivots, int* d_partition_counts, int n, int p) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        int partition = 0;
        while (partition < p - 1 && d_subarrays[tid] > d_pivots[partition]) {
            partition++;
        }
        atomicAdd(&d_partition_counts[partition], 1);
    }
}

__global__ void computeStarts(int* d_partition_counts, int* d_partition_starts, int p) {
    int tid = threadIdx.x;

    if (tid == 0) {
        int sum = 0;
        for (int i = 0; i < p; ++i) {
            d_partition_starts[i] = sum;
            sum += d_partition_counts[i];
        }
    }
}

__global__ void distributeElements(
    int* d_subarrays, int* d_output, int* d_pivots, 
    int* d_partition_starts, int* d_partition_offsets, int n, int p) 
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        int partition = 0;
        while (partition < p - 1 && d_subarrays[tid] > d_pivots[partition]) {
            partition++;
        }
        int pos = atomicAdd(&d_partition_offsets[partition], 1);
        d_output[d_partition_starts[partition] + pos] = d_subarrays[tid];
    }
}

// int main() {
//     const int n = 1e6;
//     int p = n/(1024);
//     int h_subarrays[n];
//     int h_pivots[p-1];
//     for (int i = 0; i < n; i++) {
//         h_subarrays[i] = rand() % 12715;
//     }
//     for (int i = 0; i < p-1; i++) {
//         h_pivots[i] = (i + 1) * (12715 / p);
//         std::cout<<h_pivots[i]<<" ";
//     }
//     std::cout<<"\n";

//     // Device pointers
//     int *d_subarrays, *d_output, *d_pivots, *d_partition_counts, *d_partition_starts, *d_partition_offsets;

//     // Allocate device memory
//     CUDA_CHECK(hipMalloc(&d_subarrays, n * sizeof(int)));
//     CUDA_CHECK(hipMalloc(&d_output, n * sizeof(int)));
//     CUDA_CHECK(hipMalloc(&d_pivots, (p - 1) * sizeof(int)));
//     CUDA_CHECK(hipMalloc(&d_partition_counts, p * sizeof(int)));
//     CUDA_CHECK(hipMalloc(&d_partition_starts, p * sizeof(int)));
//     CUDA_CHECK(hipMalloc(&d_partition_offsets, p * sizeof(int)));

//     // Copy data to device
//     CUDA_CHECK(hipMemcpy(d_subarrays, h_subarrays, n * sizeof(int), hipMemcpyHostToDevice));
//     CUDA_CHECK(hipMemcpy(d_pivots, h_pivots, (p - 1) * sizeof(int), hipMemcpyHostToDevice));
//     CUDA_CHECK(hipMemset(d_partition_counts, 0, p * sizeof(int)));
//     CUDA_CHECK(hipMemset(d_partition_starts, 0, p * sizeof(int)));
//     CUDA_CHECK(hipMemset(d_partition_offsets, 0, p * sizeof(int)));

//     // Kernel launch parameters
//     int blockSize = n/p;
//     int numBlocks = (n + blockSize - 1) / blockSize;

//     // Launch kernels in sequence to ensure synchronization
//     countElements<<<numBlocks, blockSize>>>(d_subarrays, d_pivots, d_partition_counts, n, p);
//     CUDA_CHECK(hipGetLastError());
//     CUDA_CHECK(hipDeviceSynchronize());

//     computeStarts<<<1, 1>>>(d_partition_counts, d_partition_starts, p);
//     CUDA_CHECK(hipGetLastError());
//     CUDA_CHECK(hipDeviceSynchronize());

//     distributeElements<<<numBlocks, blockSize>>>(d_subarrays, d_output, d_pivots, d_partition_starts, d_partition_offsets, n, p);
//     CUDA_CHECK(hipGetLastError());
//     CUDA_CHECK(hipDeviceSynchronize());

//     // Copy result back to host
//     int* h_output = new int[n];
//     CUDA_CHECK(hipMemcpy(h_output, d_output, n * sizeof(int), hipMemcpyDeviceToHost));

//     // Print result
//     for (int i = 0; i < n; ++i) {
//         std::cout << h_output[i] << " ";
//     }
//     std::cout << std::endl;

//     // Free device memory
//     CUDA_CHECK(hipFree(d_subarrays));
//     CUDA_CHECK(hipFree(d_output));
//     CUDA_CHECK(hipFree(d_pivots));
//     CUDA_CHECK(hipFree(d_partition_counts));
//     CUDA_CHECK(hipFree(d_partition_starts));
//     CUDA_CHECK(hipFree(d_partition_offsets));

//     delete[] h_output;

//     return 0;
// }


