#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <vector>

#define BLOCK_THREADS 256
#define ITEMS_PER_THREAD 1

//
// Block-sorting CUDA kernel
//
template <int BLOCK_THREADS, int ITEMS_PER_THREAD>
__global__ void BlockSortKernel(int *d_in, int *d_out)
{
    // Specialize BlockLoad, BlockStore, and BlockRadixSort collective types
    typedef hipcub::BlockLoad<
      int, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_LOAD_TRANSPOSE> BlockLoadT;
    typedef hipcub::BlockStore<
      int, BLOCK_THREADS, ITEMS_PER_THREAD, hipcub::BLOCK_STORE_TRANSPOSE> BlockStoreT;
    typedef hipcub::BlockRadixSort<
      int, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;

    // Allocate type-safe, repurposable shared memory for collectives
    __shared__ union {
        typename BlockLoadT::TempStorage       load;
        typename BlockStoreT::TempStorage      store;
        typename BlockRadixSortT::TempStorage  sort;
    } temp_storage;

    // Obtain this block's segment of consecutive keys (blocked across threads)
    int thread_keys[ITEMS_PER_THREAD];
    int block_offset = blockIdx.x * (BLOCK_THREADS * ITEMS_PER_THREAD);
    BlockLoadT(temp_storage.load).Load(d_in + block_offset, thread_keys);

    __syncthreads();        // Barrier for smem reuse

    // Collectively sort the keys
    BlockRadixSortT(temp_storage.sort).Sort(thread_keys);

    __syncthreads();        // Barrier for smem reuse

    // Store the sorted segment
    BlockStoreT(temp_storage.store).Store(d_out + block_offset, thread_keys);
}

int main() {
    // Initialize host data
    std::vector<int> h_data = { 34, 78, 12, 56, 89, 21, 90, 34, 23, 45, 67, 11, 23, 56, 78, 99, 123, 45, 67, 89, 23, 45, 67, 34, 78 };
    int n = h_data.size();

    // Allocate device memory
    int* d_data;
    hipMalloc(&d_data, n * sizeof(int));
    int* d_sorted_data;
    hipMalloc(&d_sorted_data, n * sizeof(int));

    // Copy data to device
    hipMemcpy(d_data, h_data.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = (n + (BLOCK_SIZE * ITEMS_PER_THREAD) - 1) / (BLOCK_SIZE * ITEMS_PER_THREAD);

    // Launch kernel to sort blocks
    BlockSortKernel<BLOCK_SIZE, ITEMS_PER_THREAD><<<numBlocks, BLOCK_SIZE>>>(d_data, d_sorted_data);

    // Copy sorted data back to host
    hipMemcpy(h_data.data(), d_sorted_data, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print sorted blocks
    for (int i = 0; i < h_data.size(); i++) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_data);
    hipFree(d_sorted_data);

    return 0;
}
