#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <vector>

#define BLOCK_SIZE 256

// Kernel to sort each block individually using CUB's radix sort
__global__ void sortBlocks(int* d_data, int n) {
    int offset = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    // Allocate shared memory for sorting within the block
    extern __shared__ int sharedData[];

    // Load data into shared memory
    if (offset < n) {
        sharedData[threadIdx.x] = d_data[offset];
    } 
    else {
        sharedData[threadIdx.x] = INT_MAX;
    }
    __syncthreads();

    // Sorting within the block using CUB
    typedef hipcub::BlockRadixSort<int, BLOCK_SIZE> BlockRadixSort;
    __shared__ typename BlockRadixSort::TempStorage temp_storage;

    BlockRadixSort(temp_storage).Sort(sharedData);

    __syncthreads();

    // Write sorted data back to global memory
    if (offset < n) {
        d_data[offset] = sharedData[threadIdx.x];
    }
}

int main() {
    // Initialize host data
    std::vector<int> h_data = { 34, 78, 12, 56, 89, 21, 90, 34, 23, 45, 67, 11, 23, 56, 78, 99, 123, 45, 67, 89, 23, 45, 67, 34, 78 };
    int n = h_data.size();

    // Allocate device memory
    int* d_data;
    hipMalloc(&d_data, n * sizeof(int));

    // Copy data to device
    hipMemcpy(d_data, h_data.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Launch kernel to sort blocks
    sortBlocks<<<numBlocks, BLOCK_SIZE, BLOCK_SIZE * sizeof(int)>>>(d_data, n);

    // Copy sorted data back to host
    hipMemcpy(h_data.data(), d_data, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print sorted blocks
    for (int i = 0; i < h_data.size(); i++) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_data);

    return 0;
}


// Data Initialization: We initialize the data on the host and copy it to the device.\

// Kernel Launch: We launch a kernel with a number of blocks that covers the entire dataset. Each block operates on a chunk of the data.

// Shared Memory and Synchronization: Each block loads its chunk of data into shared memory. This local shared memory is used to avoid race conditions.

// Radix Sort: We implement the radix sort within each block, ensuring thread synchronization using __syncthreads().

// Copy Back to Host: After sorting, the data is copied back to the host and printed.


