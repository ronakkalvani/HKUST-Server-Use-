#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// Error handling macro
#define CUDA_CHECK(call)                                                       \
    do {                                                                       \
        hipError_t error = call;                                              \
        if (error != hipSuccess) {                                            \
            std::cerr << "CUDA Error: " << hipGetErrorString(error)           \
                      << " at " << __FILE__ << ":" << __LINE__ << std::endl;   \
            exit(error);                                                       \
        }                                                                      \
    } while (0)

// Kernel to initialize flags based on new segment starts
__global__ void InitializeFlags(const int* input, int* flags, int num_items) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_items) {
        flags[idx] = (idx == 0 || input[idx] != input[idx - 1]) ? 1 : 0;
    }
}

void SegmentedPrefixSum(const std::vector<int>& input, std::vector<int>& output) {
    const int num_items = input.size();

    int* d_input = nullptr;
    int* d_output = nullptr;
    int* d_flags = nullptr;
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_input, num_items * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_output, num_items * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_flags, num_items * sizeof(int)));

    // Copy input data to device
    CUDA_CHECK(hipMemcpy(d_input, input.data(), num_items * sizeof(int), hipMemcpyHostToDevice));

    // Initialize flags
    const int block_size = 256;
    const int grid_size = (num_items + block_size - 1) / block_size;
    InitializeFlags<<<grid_size, block_size>>>(d_input, d_flags, num_items);
    CUDA_CHECK(hipDeviceSynchronize());

    // Determine temporary device storage requirements
    hipcub::DeviceSegmentedScan::InclusiveSum(
        d_temp_storage, temp_storage_bytes,
        d_input, d_output,
        num_items,
        num_items,  // number of segments
        d_flags, d_flags + 1);
    
    // Allocate temporary storage
    CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));

    // Run segmented prefix sum
    hipcub::DeviceSegmentedScan::InclusiveSum(
        d_temp_storage, temp_storage_bytes,
        d_input, d_output,
        num_items,
        num_items,  // number of segments
        d_flags, d_flags + 1);

    // Copy results back to host
    CUDA_CHECK(hipMemcpy(output.data(), d_output, num_items * sizeof(int), hipMemcpyDeviceToHost));

    // Free device memory
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_flags));
    CUDA_CHECK(hipFree(d_temp_storage));
}

int main() {
    std::vector<int> input = {1, 2, 2, 3, 1, 1, 2, 3, 3, 3};
    std::vector<int> output(input.size(), 0);

    SegmentedPrefixSum(input, output);

    // Print the result
    for (int val : output) {
        std::cout << val << " ";
    }
    std::cout << std::endl;

    return 0;
}
