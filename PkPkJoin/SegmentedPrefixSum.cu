#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>

__global__ void segmentedPrefixSum(int *input, int *output, int n, int blockSize) {
    extern __shared__ int shared[];

    int tid = threadIdx.x;
    int global_tid = blockIdx.x * blockDim.x + tid;

    // Load input into shared memory
    if (global_tid < n) {
        shared[tid] = input[global_tid];
    }
    __syncthreads();

    // Initialize prefix sum within segments
    if (global_tid < n) {
        if (tid == 0 || shared[tid] != shared[tid - 1]) {
            output[global_tid] = 0;
        } else {
            output[global_tid] = output[global_tid - 1] + 1;
        }
    }

    // Perform prefix sum within segments in shared memory
    for (int stride = 1; stride < blockSize; stride *= 2) {
        __syncthreads();
        if (tid >= stride && shared[tid] == shared[tid - stride]) {
            output[global_tid] += output[global_tid - stride];
        }
    }
}

// int main() {
//     int blockSize = 512;
//     int n = 1e6;  // Large dataset size
//     std::vector<int> h_input(n);

//     for (int i = 0; i < n; ++i) {
//         h_input[i] = i / 6;
//     }

//     int *d_input, *d_output;
//     cudaMalloc(&d_input, n * sizeof(int));
//     cudaMalloc(&d_output, n * sizeof(int));

//     cudaMemcpy(d_input, h_input.data(), n * sizeof(int), cudaMemcpyHostToDevice);

//     int numBlocks = (n + blockSize - 1) / blockSize;
//     segmentedPrefixSum<<<numBlocks, blockSize, blockSize * sizeof(int)>>>(d_input, d_output, n, blockSize);

//     std::vector<int> h_output(n);
//     cudaMemcpy(h_output.data(), d_output, n * sizeof(int), cudaMemcpyDeviceToHost);

//     // Print input and output for verification
//     std::cout << "Input:" << std::endl;
//     for (int i = 0; i < n; ++i) {
//         std::cout << h_input[i] << " ";
//         if ((i + 1) % blockSize == 0) std::cout << std::endl;
//     }
//     std::cout << std::endl;

//     std::cout << "Output:" << std::endl;
//     for (int i = 0; i < n; ++i) {
//         std::cout << h_output[i] << " ";
//         if ((i + 1) % blockSize == 0) std::cout << std::endl;
//     }

//     cudaFree(d_input);
//     cudaFree(d_output);

//     return 0;
// }
