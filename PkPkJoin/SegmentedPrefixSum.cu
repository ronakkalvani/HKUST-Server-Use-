#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>

__global__ void segmentedPrefixSum(int *input, int *output, int n, int blockSize) {
    extern __shared__ int shared[];

    int tid = threadIdx.x;
    int global_tid = blockIdx.x * blockDim.x + tid;

    // Load input into shared memory
    if (global_tid < n) {
        shared[tid] = input[global_tid];
    } else {
        shared[tid] = -1;  // Set to a value that won't match any segment
    }
    __syncthreads();

    // Perform prefix sum within segments in shared memory
    for (int stride = 1; stride < blockSize; stride *= 2) {
        int temp;
        if (tid >= stride && shared[tid] == shared[tid - stride]) {
            temp = output[global_tid - stride] + 1;
        } else {
            temp = 0;
        }
        __syncthreads(); // Ensure all threads have computed their temporary values
        if (tid >= stride && shared[tid] == shared[tid - stride]) {
            output[global_tid] += temp;
        }
        __syncthreads();
    }

    // Write final values to the output array
    if (tid > 0 && shared[tid] == shared[tid - 1]) {
        output[global_tid] += output[global_tid - 1] + 1;
    } else {
        output[global_tid] = 0;
    }
}

int main() {
    int blockSize = 512;
    int n = 10000;  // Large dataset size
    std::vector<int> h_input(n);

    for (int i = 0; i < n; ++i) {
        h_input[i] = i / 100;  // Example initialization, adjust as needed
    }

    int *d_input, *d_output;
    hipMalloc(&d_input, n * sizeof(int));
    hipMalloc(&d_output, n * sizeof(int));

    hipMemcpy(d_input, h_input.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = (n + blockSize - 1) / blockSize;
    segmentedPrefixSum<<<numBlocks, blockSize, blockSize * sizeof(int)>>>(d_input, d_output, n, blockSize);

    std::vector<int> h_output(n);
    hipMemcpy(h_output.data(), d_output, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print input and output for verification
    std::cout << "Input:" << std::endl;
    for (int i = 0; i < n; ++i) {
        std::cout << h_input[i] << " ";
        if ((i + 1) % blockSize == 0) std::cout << std::endl;
    }
    std::cout << std::endl;

    std::cout << "Output:" << std::endl;
    for (int i = 0; i < n; ++i) {
        std::cout << h_output[i] << " ";
        if ((i + 1) % blockSize == 0) std::cout << std::endl;
    }

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}


// #include <cuda_runtime.h>
// #include <iostream>
// #include <vector>
// #include <cstdlib>
// #include <ctime>

// __global__ void segmentedPrefixSum(int *input, int *output, int n, int blockSize) {
//     extern __shared__ int shared[];

//     int tid = threadIdx.x;
//     int global_tid = blockIdx.x * blockDim.x + tid;

//     // Load input into shared memory
//     if (global_tid < n) {
//         shared[tid] = input[global_tid];
//     }
//     __syncthreads();

//     // Initialize prefix sum within segments
//     if (global_tid < n) {
//         if (tid == 0 || shared[tid] != shared[tid - 1]) {
//             output[global_tid] = 0;
//         } else {
//             output[global_tid] = output[global_tid - 1] + 1;
//         }
//     }

//     // Perform prefix sum within segments in shared memory
//     for (int stride = 1; stride < blockSize; stride *= 2) {
//         __syncthreads();
//         if (tid >= stride && shared[tid] == shared[tid - stride]) {
//             output[global_tid] += output[global_tid - stride];
//         }
//     }
// }

// int main() {
//     int blockSize = 1024;
//     int n = 1e4;  // Large dataset size
//     std::vector<int> h_input(n);

//     for (int i = 0; i < n; ++i) {
//         if (i<100) h_input[i] = i /10;
//         else h_input[i] = i /100;
//     }

//     // int blockSize = 8;
//     // std::vector<int> h_input = {
//     //     0, 0, 0, 0, 0, 1, 1, 1,
//     //     0, 0, 0, 1, 1, 1, 1, 1,
//     //     0, 0, 1, 1, 1, 1, 1, 1
//     // };
//     // int n = h_input.size();

//     int *d_input, *d_output;
//     cudaMalloc(&d_input, n * sizeof(int));
//     cudaMalloc(&d_output, n * sizeof(int));

//     cudaMemcpy(d_input, h_input.data(), n * sizeof(int), cudaMemcpyHostToDevice);

//     int numBlocks = (n + blockSize - 1) / blockSize;
//     segmentedPrefixSum<<<numBlocks, blockSize, blockSize * sizeof(int)>>>(d_input, d_output, n, blockSize);

//     std::vector<int> h_output(n);
//     cudaMemcpy(h_output.data(), d_output, n * sizeof(int), cudaMemcpyDeviceToHost);

//     // Print input and output for verification
//     std::cout << "Input:" << std::endl;
//     for (int i = 0; i < n; ++i) {
//         std::cout << h_input[i] << " ";
//         if ((i + 1) % blockSize == 0) std::cout << std::endl;
//     }
//     std::cout << std::endl;

//     std::cout << "Output:" << std::endl;
//     for (int i = 0; i < n; ++i) {
//         std::cout << h_output[i] << " ";
//         if ((i + 1) % blockSize == 0) std::cout << std::endl;
//     }

//     cudaFree(d_input);
//     cudaFree(d_output);

//     return 0;
// }
