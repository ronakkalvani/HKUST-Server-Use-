#include "hip/hip_runtime.h"
#include <iostream>
#include <hipcub/hipcub.hpp>

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " code=" << err << " \"" << hipGetErrorString(err) << "\"" << std::endl; \
            exit(1); \
        } \
    } while (0)

// Function to print array
template <typename T>
void PrintArray(const T* array, int size, const char* label) {
    std::cout << label << ": ";
    for (int i = 0; i < size; ++i) {
        std::cout << array[i] << " ";
    }
    std::cout << std::endl;
}

__global__ void InitFlags(const int* d_input, int* d_flags, int num_items) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_items) {
        if (tid == 0) {
            d_flags[tid] = 1;
        } else {
            d_flags[tid] = (d_input[tid] != d_input[tid - 1]) ? 1 : 0;
        }
    }
}

int main() {
    const int num_items = 10;
    int h_input[num_items] = {1, 1, 1, 2, 2, 1, 1, 3, 3, 1};
    int h_output[num_items];

    int* d_input = nullptr;
    int* d_output = nullptr;
    int* d_flags = nullptr;
    int* d_segment_offsets = nullptr;
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;

    CUDA_CHECK(hipMalloc((void**)&d_input, num_items * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_output, num_items * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_flags, num_items * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_segment_offsets, (num_items + 1) * sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_input, h_input, num_items * sizeof(int), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (num_items + threads - 1) / threads;

    InitFlags<<<blocks, threads>>>(d_input, d_flags, num_items);
    CUDA_CHECK(hipDeviceSynchronize());

    // Compute the segment offsets
    hipcub::DeviceScan::InclusiveSum(nullptr, temp_storage_bytes, d_flags, d_segment_offsets + 1, num_items);
    CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_flags, d_segment_offsets + 1, num_items);

    // Set the first element of segment_offsets to 0
    CUDA_CHECK(hipMemset(d_segment_offsets, 0, sizeof(int)));

    // Perform the segmented prefix sum
    hipcub::DeviceSegmentedScan::InclusiveSum(nullptr, temp_storage_bytes, d_input, d_output, num_items, num_items, d_segment_offsets, d_segment_offsets + 1);
    CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));
    hipcub::DeviceSegmentedScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_input, d_output, num_items, num_items, d_segment_offsets, d_segment_offsets + 1);

    CUDA_CHECK(hipMemcpy(h_output, d_output, num_items * sizeof(int), hipMemcpyDeviceToHost));

    PrintArray(h_input, num_items, "Input");
    PrintArray(h_output, num_items, "Output");

    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_flags));
    CUDA_CHECK(hipFree(d_segment_offsets));
    CUDA_CHECK(hipFree(d_temp_storage));

    return 0;
}
