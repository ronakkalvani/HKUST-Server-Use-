#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <hiprand/hiprand_kernel.h>
#include <cmath>

// #define BLOCK_THREADS 32

#define CUDA_CHECK(call)                                                   \
    do {                                                                   \
        hipError_t error = call;                                          \
        if (error != hipSuccess) {                                        \
            std::cerr << "CUDA Error: " << hipGetErrorString(error) <<    \
            " at " << __FILE__ << ":" << __LINE__ << std::endl;            \
            exit(1);                                                       \
        }                                                                  \
    } while (0)

__global__ void Splitterss(int* d_splitters,int* d_samples,int sample_size,int p) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // if (tid < p-1) {
    //     d_splitters[tid] = d_samples[(tid + 1) * sample_size / p];
    // }
    if (tid==0) {
        for(int i=0;i<p-1;i++) {
            d_splitters[i] = d_samples[(i + 1) * sample_size / p];
        }
    }
}

// Kernel to initialize the CURAND states
__global__ void initCurand(hiprandState* state, unsigned long seed, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        hiprand_init(seed, tid, 0, &state[tid]);
    }
}

// Kernel to sample elements
__global__ void sampleElements(hiprandState* state, int* d_sorted_subarrays, int* d_samples, int n, int sample_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < sample_size) {
        int ind = hiprand(&state[tid]) % n;
        d_samples[tid] = d_sorted_subarrays[ind];
    }
}

void FindSplit(int* d_sorted_data, int* d_samples, int* d_splitters, int n, int p, int sample_size, hiprandState* d_state) {
    int blockSize = BLOCK_THREADS;
    int numBlocks = (sample_size + blockSize - 1) / blockSize;

    sampleElements<<<numBlocks, blockSize>>>(d_state, d_sorted_data, d_samples, n, sample_size);
    hipDeviceSynchronize();
    
    // Sort samples using CUB
    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    
    // Determine temporary device storage requirements
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_samples, d_samples, sample_size);
    CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));
    
    // Run sorting operation
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_samples, d_samples, sample_size);
    
    // Free temporary storage
    hipFree(d_temp_storage);
}

// int main() {
//     const int n = 1e6;
//     int p = 1e4;
//     int sample_size = p*int(log2(p));

//     int h_sorted_subarrays[n];
//     for (int i = 0; i < n; i++) {
//         h_sorted_subarrays[i] = rand() % 123556;
//     }
//     int h_splitters[p - 1];

//     int *d_sorted_subarrays, *d_samples, *d_splitters;
//     hiprandState* d_state;

//     CUDA_CHECK(hipMalloc(&d_sorted_subarrays, n * sizeof(int)));
//     CUDA_CHECK(hipMalloc(&d_samples, sample_size * sizeof(int)));
//     CUDA_CHECK(hipMalloc(&d_splitters, (p - 1) * sizeof(int)));
//     CUDA_CHECK(hipMalloc(&d_state, sample_size * sizeof(hiprandState)));

//     CUDA_CHECK(hipMemcpy(d_sorted_subarrays, h_sorted_subarrays, n * sizeof(int), hipMemcpyHostToDevice));

//     int blockSize = BLOCK_THREADS;
//     int numBlocks = (sample_size + blockSize - 1) / blockSize;
//     initCurand<<<numBlocks, blockSize>>>(d_state, time(NULL), sample_size);
//     CUDA_CHECK(hipGetLastError());
//     CUDA_CHECK(hipDeviceSynchronize());

//     FindSplit(d_sorted_subarrays, d_samples, d_splitters, n, p, sample_size, d_state);

//     int* h_samples = new int[sample_size];
//     CUDA_CHECK(hipMemcpy(h_samples, d_samples, sample_size * sizeof(int), hipMemcpyDeviceToHost));

//     for (int i = 0; i < p - 1; ++i) {
//         h_splitters[i] = h_samples[(i + 1) * sample_size / p];
//     }

//     CUDA_CHECK(hipFree(d_sorted_subarrays));
//     CUDA_CHECK(hipFree(d_samples));
//     CUDA_CHECK(hipFree(d_splitters));
//     CUDA_CHECK(hipFree(d_state));

//     delete[] h_samples;

//     for (int i = 0; i < p - 1; ++i) {
//         std::cout << "Splitter " << i << ": " << h_splitters[i] << std::endl;
//     }

//     return 0;
// }


// #include <iostream>
// #include <vector>
// #include <algorithm>
// #include <hip/hip_runtime.h>
// #include <hipcub/hipcub.hpp>
// #include <hiprand/hiprand_kernel.h>

// #define BLOCK_THREADS 32


// #define CUDA_CHECK(call)                                                   \
//     do {                                                                   \
//         hipError_t error = call;                                          \
//         if (error != hipSuccess) {                                        \
//             std::cerr << "CUDA Error: " << hipGetErrorString(error) <<    \
//             " at " << __FILE__ << ":" << __LINE__ << std::endl;            \
//             exit(1);                                                       \
//         }                                                                  \
//     } while (0)

// __global__ void Splitterss(int* d_splitters,int* d_samples,int sample_size,int p) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     if (tid == 0) {
//         for (int i = 0; i < p - 1; ++i) {
//             d_splitters[i] = d_samples[(i + 1) * sample_size / p];
//         }
//     }
// }

// // Kernel to sample elements
// __global__ void sampleElements(int* d_sorted_subarrays, int* d_samples, int n, int sample_size, int stride) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     if (tid < sample_size) {
//         int ind=rand()%n;
//         d_samples[tid] = d_sorted_subarrays[ind];
//     }
// }

// void FindSplit(int* d_sorted_data, int* d_samples, int* d_splitters, int n, int p,int sample_size) {
//     int blockSize = BLOCK_THREADS;
//     int numBlocks = (sample_size + blockSize - 1) / blockSize;
//     int stride = n / sample_size;

//     sampleElements<<<numBlocks, blockSize>>>(d_sorted_data, d_samples, n, sample_size, stride);
//     hipDeviceSynchronize();
    
//     // Sort samples using CUB
//     void* d_temp_storage = nullptr;
//     size_t temp_storage_bytes = 0;
    
//     // Determine temporary device storage requirements
//     hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_samples, d_samples, sample_size);
//     CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));
    
//     // Run sorting operation
//     hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_samples, d_samples, sample_size);
    
//     // Free temporary storage
//     hipFree(d_temp_storage);
// }

// int main() {
//     // Example data
//     const int n = 256;
//     int p = 8;
//     int sample_size = n/p; // Adjust sample size as needed
//     // int h_sorted_subarrays[] = {1, 3, 5, 7, 9, 11, 13, 15, 17, 19, 21, 23, 25, 27, 29, 31};
//     int h_sorted_subarrays[n];
//     for (int i=0;i<n;i++) {
//         h_sorted_subarrays[i] = rand() % 123;
//     }
//     int h_splitters[p - 1];

//     // Device pointers
//     int *d_sorted_subarrays, *d_samples, *d_splitters;
    
//     // Allocate device memory
//     CUDA_CHECK(hipMalloc(&d_sorted_subarrays, n * sizeof(int)));
//     CUDA_CHECK(hipMalloc(&d_samples, sample_size * sizeof(int)));
//     CUDA_CHECK(hipMalloc(&d_splitters, (p - 1) * sizeof(int)));
    
//     // Copy data to device
//     CUDA_CHECK(hipMemcpy(d_sorted_subarrays, h_sorted_subarrays, n * sizeof(int), hipMemcpyHostToDevice));
    
//     // Launch kernel to sample elements
//     int blockSize = n/p;
//     int numBlocks = (sample_size + blockSize - 1) / blockSize;
//     int stride = n / sample_size;
//     sampleElements<<<numBlocks, blockSize>>>(d_sorted_subarrays, d_samples, n, sample_size, stride);
//     CUDA_CHECK(hipGetLastError());
//     CUDA_CHECK(hipDeviceSynchronize());
    
//     // Sort samples using CUB
//     void* d_temp_storage = nullptr;
//     size_t temp_storage_bytes = 0;
    
//     // Determine temporary device storage requirements
//     hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_samples, d_samples, sample_size);
//     CUDA_CHECK(hipMalloc(&d_temp_storage, temp_storage_bytes));
    
//     // Run sorting operation
//     hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_samples, d_samples, sample_size);
    
//     // Free temporary storage
//     CUDA_CHECK(hipFree(d_temp_storage));
    
//     // Select splitters
//     int* h_samples = new int[sample_size];
//     CUDA_CHECK(hipMemcpy(h_samples, d_samples, sample_size * sizeof(int), hipMemcpyDeviceToHost));
    
//     for (int i = 0; i < p - 1; ++i) {
//         h_splitters[i] = h_samples[(i + 1) * sample_size / p];
//     }
    
//     // Free device memory
//     CUDA_CHECK(hipFree(d_sorted_subarrays));
//     CUDA_CHECK(hipFree(d_samples));
//     CUDA_CHECK(hipFree(d_splitters));
    
//     delete[] h_samples;
    
//     // Print splitters
//     for (int i = 0; i < p - 1; ++i) {
//         std::cout << "Splitter " << i << ": " << h_splitters[i] << std::endl;
//     }

//     return 0;
// }
