#include <iostream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/SortDataBlockWise.cu"
#include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/FindSplits.cu"
#include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/DistributionAfterSplits.cu"
// #include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/JoinAfterSort.cu"


int main() {
    std::vector<int> h_data(100);
    for (int i=0;i<h_data.size();i++) {
        h_data[i]=rand() % 37;
    }
    int n = h_data.size();

    // Allocate device memory
    int* d_data;
    hipMalloc(&d_data, n * sizeof(int));
    int* d_sorted_data;
    hipMalloc(&d_sorted_data, n * sizeof(int));

    // Copy data to device
    hipMemcpy(d_data, h_data.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = (n + (BLOCK_THREADS * ITEMS_PER_THREAD) - 1) / (BLOCK_THREADS * ITEMS_PER_THREAD);

    // Launch kernel to sort blocks
    BlockSortKernel<<<numBlocks, BLOCK_THREADS>>>(d_data, d_sorted_data, n);

    int p = numBlocks;
    int sample_size = n / p;
    int *d_samples, *d_splitters;
    hipMalloc(&d_samples, sample_size * sizeof(int));
    hipMalloc(&d_splitters, (p - 1) * sizeof(int));

    FindSplit(d_sorted_data,d_samples, d_splitters, n, numBlocks, sample_size);

    int *d_output,*d_partition_counts;

    // Allocate device memory

    CUDA_CHECK(hipMalloc(&d_output, n * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_partition_counts, p * sizeof(int)));

    // Copy data to device
    CUDA_CHECK(hipMemset(d_partition_counts, 0, p * sizeof(int)));

    int blockSize = numBlocks;

    // Launch kernel to merge partitions
    mergePartitions<<<numBlocks, blockSize>>>(d_sorted_data, d_partition_counts, d_output, d_samples, d_partition_counts, n, p);
    CUDA_CHECK(hipDeviceSynchronize());

    // Copy result back to host
    int* h_output = new int[n];
    CUDA_CHECK(hipMemcpy(h_output, d_output, n * sizeof(int), hipMemcpyDeviceToHost));

    // Print result
    for (int i = 0; i < n; ++i) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    delete[] h_output;

    // Free device memory
    hipFree(d_data);
    hipFree(d_sorted_data);
    hipFree(d_samples);
    hipFree(d_splitters);
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_partition_counts));

    return 0;
}