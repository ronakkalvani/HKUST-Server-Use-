#include <iostream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/DistributionAfterSplits.cu"
#include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/FindSplits.cu"
#include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/JoinAfterSort.cu"
#include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/SortDataBlockWise.cu"

#define BLOCK_THREADS 16
#define ITEMS_PER_THREAD 1

int main() {
    std::vector<int> h_data(100);
    for (int i=0;i<h_data.size();i++) {
        h_data[i]=rand()%7;
    }
    int n = h_data.size();

    // Allocate device memory
    int* d_data;
    hipMalloc(&d_data, n * sizeof(int));
    int* d_sorted_data;
    hipMalloc(&d_sorted_data, n * sizeof(int));

    // Copy data to device
    hipMemcpy(d_data, h_data.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = (n + (BLOCK_THREADS * ITEMS_PER_THREAD) - 1) / (BLOCK_THREADS * ITEMS_PER_THREAD);

    // Launch kernel to sort blocks
    BlockSortKernel<<<numBlocks, BLOCK_THREADS>>>(d_data, d_sorted_data, n);

    // Copy sorted data back to host
    hipMemcpy(h_data.data(), d_sorted_data, n * sizeof(int), hipMemcpyDeviceToHost);

    // Print sorted blocks
    for (int i = 0; i < h_data.size(); i++) {
        std::cout << h_data[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_data);
    hipFree(d_sorted_data);

    return 0;
}