#include <iostream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define BLOCK_THREADS 512
#define ITEMS_PER_THREAD 1
#define BLOCK_THREAD 2*BLOCK_THREADS

#include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/SortDataBlockWise.cu"
#include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/FindSplits.cu"
#include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/DistributionAfterSplits.cu"
#include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/FinalSorting.cu"
// #include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/JoinAfterSort.cu"


int main() {
    std::vector<int> h_data(1e8);
    for (int i=0;i<h_data.size();i++) {
        h_data[i]=rand()%1257245;
    }
    int n = h_data.size();

    // Allocate device memory
    int* d_data;
    hipMalloc(&d_data, n * sizeof(int));
    int* d_sorted_data;
    hipMalloc(&d_sorted_data, n * sizeof(int));

    // Copy data to device
    hipMemcpy(d_data, h_data.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = (n + (BLOCK_THREADS * ITEMS_PER_THREAD) - 1) / (BLOCK_THREADS * ITEMS_PER_THREAD);

    // Launch kernel to sort blocks
    BlockSortKernel<<<numBlocks, BLOCK_THREADS>>>(d_data, d_sorted_data, n);

    int p = numBlocks;
    int sample_size = n/p;
    int *d_samples, *d_splitters;
    hipMalloc(&d_samples, sample_size * sizeof(int));
    hipMalloc(&d_splitters, (p - 1) * sizeof(int));

    FindSplit(d_sorted_data,d_samples, d_splitters, n, numBlocks, sample_size);

    Splitterss<<<1,1>>> (d_splitters,d_samples,sample_size,p);
    hipDeviceSynchronize();
    // printArray<<<1,1>>> (d_splitters,p-1);
    // hipDeviceSynchronize();

    int blockSize = BLOCK_THREADS;
    // Device pointers
    int  *d_output, *d_partition_counts, *d_partition_starts, *d_partition_offsets;

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_output, n * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_partition_counts, p * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_partition_starts, p * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_partition_offsets, p * sizeof(int)));

    // Copy data to device
    CUDA_CHECK(hipMemset(d_partition_counts, 0, p * sizeof(int)));
    CUDA_CHECK(hipMemset(d_partition_starts, 0, p * sizeof(int)));
    CUDA_CHECK(hipMemset(d_partition_offsets, 0, p * sizeof(int)));

    // Launch kernels in sequence to ensure synchronization
    countElements<<<numBlocks, blockSize>>>(d_sorted_data, d_splitters, d_partition_counts, n, p);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    computeStarts<<<1, 1>>>(d_partition_counts, d_partition_starts, p);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    distributeElements<<<numBlocks, blockSize>>>(d_sorted_data, d_output, d_splitters, d_partition_starts, d_partition_offsets, n, p);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    // printArray<<<1,1>>>(d_output,10000);
    // CUDA_CHECK(hipGetLastError());
    // CUDA_CHECK(hipDeviceSynchronize());

    // printArray<<<1,1>>>(d_partition_starts,p);
    // CUDA_CHECK(hipGetLastError());
    // CUDA_CHECK(hipDeviceSynchronize());
    

    int* d_final_array;
    hipMalloc(&d_final_array, n * sizeof(int));
    // BlockSortKernel<<<numBlocks, BLOCK_THREADS>>>(d_output, d_final_array,n);
    BlockSortKernel2<<<numBlocks, BLOCK_THREAD>>>(d_output, d_final_array, d_partition_starts,p,n);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    printArray<<<1,1>>>(d_final_array,10000);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());


    // Free device memory
    hipFree(d_data);
    hipFree(d_sorted_data);
    hipFree(d_samples);
    hipFree(d_splitters);
    CUDA_CHECK(hipFree(d_output));
    CUDA_CHECK(hipFree(d_partition_starts));
    CUDA_CHECK(hipFree(d_partition_offsets));
    CUDA_CHECK(hipFree(d_partition_counts));

    return 0;
}