#include <iostream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/SortDataBlockWise.cu"
#include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/FindSplits.cu"
// #include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/DistributionAfterSplits.cu"
// #include "/csproject/yike/intern/ronak/HKUST-Server-Use-/PkPkJoin/Ronak/JoinAfterSort.cu"


int main() {
    std::vector<int> h_data(100);
    for (int i=0;i<h_data.size();i++) {
        h_data[i]=rand() % 37;
    }
    int n = h_data.size();

    // Allocate device memory
    int* d_data;
    hipMalloc(&d_data, n * sizeof(int));
    int* d_sorted_data;
    hipMalloc(&d_sorted_data, n * sizeof(int));

    // Copy data to device
    hipMemcpy(d_data, h_data.data(), n * sizeof(int), hipMemcpyHostToDevice);

    int numBlocks = (n + (BLOCK_THREADS * ITEMS_PER_THREAD) - 1) / (BLOCK_THREADS * ITEMS_PER_THREAD);

    // Launch kernel to sort blocks
    BlockSortKernel<<<numBlocks, BLOCK_THREADS>>>(d_data, d_sorted_data, n);

    int p = numBlocks;
    int sample_size = n / p;
    int *d_samples, *d_splitters;
    hipMalloc(&d_samples, sample_size * sizeof(int));
    hipMalloc(&d_splitters, (p - 1) * sizeof(int));

    FindSplit(d_sorted_data,d_samples, d_splitters, n, numBlocks, sample_size);

    // Select splitters
    int* h_samples = new int[sample_size];
    CUDA_CHECK(hipMemcpy(h_samples, d_samples, sample_size * sizeof(int), hipMemcpyDeviceToHost));
    
    int h_splitters[p - 1];
    for (int i = 0; i < p - 1; ++i) {
        h_splitters[i] = h_samples[(i + 1) * sample_size / p];
    }
    
    delete[] h_samples;
    
    // Print splitters
    for (int i = 0; i < p - 1; ++i) {
        std::cout << "Splitter " << i << ": " << h_splitters[i] << std::endl;
    }

    // Free device memory
    hipFree(d_data);
    hipFree(d_sorted_data);
    hipFree(d_samples);
    hipFree(d_splitters);

    return 0;
}