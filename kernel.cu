﻿
#include "hip/hip_runtime.h"



#include <stdio.h>

__global__ void cuda_hello() {
    printf("Hello World from GPU!\n");
}

int main() {
    printf("Hello World from CPU!\n");
    cuda_hello<<<1,1>>>();
    hipDeviceSynchronize();
    return 0;
}


