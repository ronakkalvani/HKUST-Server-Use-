#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <iostream>

// Kernel function to print the sorted data
__global__ void print_sorted_data(int* device_data, int num_items)
{
    if (threadIdx.x == 0)
    {
        printf("Sorted data: ");
        for (int i = 0; i < num_items; i++)
            printf("%d ", device_data[i]);
        printf("\n");
    }
}

int main()
{
    // Initialize host data
    int h_data[] = {1, 5, 2, 4, 3};
    int num_items = sizeof(h_data) / sizeof(h_data[0]);

    // Initialize device data
    int* d_data;
    hipMalloc(&d_data, sizeof(h_data));
    hipMemcpy(d_data, h_data, sizeof(h_data), hipMemcpyHostToDevice);

    // Allocate device memory for sorted data
    int* d_sorted_data;
    hipMalloc(&d_sorted_data, sizeof(h_data));

    // Allocate temporary storage
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_data, d_sorted_data, num_items);
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Sort data
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_data, d_sorted_data, num_items);

    // Print sorted data
    print_sorted_data<<<1, 1>>>(d_sorted_data, num_items);
    hipDeviceSynchronize();
    
    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; // add
    hipProfilerStop();
    return 0;

    // Cleanup
    hipFree(d_data);
    hipFree(d_sorted_data);
    hipFree(d_temp_storage);

    return 0;
}

